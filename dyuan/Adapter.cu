
#include <hip/hip_runtime.h>
// #include "Adapter.cuh"

// void Adapter::EMOp1()
// {
//     theEMFields_Cyl3D->m_FldSrcs_Cyl3D->Advance();
//     theEMFields_Cyl3D->m_FldSrcs_Cyl3D->Advance_SI_MJ(1.0);
//     theEMFields_Cyl3D->m_FldSrcs_Cyl3D->Advance_SI_Mag_Damping_0(1.0);

//     void (*funcPtr)() = &g_AdvanceMagCntr_WithDamping_m_Mphi;
//     std::cout << reinterpret_cast<void *>(funcPtr) << std::endl;
// }

// void Adapter::EMGraph1()
// {
//     cudaStreamBeginCapture(graphs->computeStream, cudaStreamCaptureModeGlobal);

//     dim3 block(256);
//     dim3 grid((unsigned int)ceil(theEMFields_Cyl3D->m_MphiDatasNum / (float)block.x));

//     g_AdvanceMagCntr_WithDamping_m_Mphi<<<grid, block, 0, graphs->computeStream>>>(
//         theEMFields_Cyl3D->m_h_d_MphiFuncArray_Cyl3D, theEMFields_Cyl3D->m_MphiDatasNum, m_h_d_MphiDatasPtr, m_h_d_EzrDatasPtr);

//     block.x = 256;
//     grid.x = ((unsigned int)ceil(theEMFields_Cyl3D->m_MzrDatasNum / (float)block.x));

//     g_AdvanceMagCntr_WithDamping_m_Mzr<<<grid, block, 0, graphs->computeStream>>>(
//         theEMFields_Cyl3D->m_h_d_MzrFuncArray_Cyl3D, theEMFields_Cyl3D->m_MzrDatasNum, m_h_d_MzrDatasPtr, m_h_d_EphiDatasPtr, m_h_d_EzrDatasPtr);

//     cudaStreamEndCapture(graphs->computeStream, &graphs->EMGraph1);
//     cudaGraphInstantiate(&graphs->EMGraphExec1, graph->EMGraph1, NULL, NULL, 0);
// }

// void Adapter::EMOp2()
// {
//     theEMFields_Cyl3D->m_FldSrcs_Cyl3D->Advance_SI_Mag_Damping_1(1.0);
//     theEMFields_Cyl3D->m_FldSrcs_Cyl3D->Advance_SI_J(1.0);
//     theEMFields_Cyl3D->m_FldSrcs_Cyl3D->Advance_SI_Elec_Damping_0(1.0, theEMFields_Cyl3D->m_Damping);
// }

// Adapter::~Adapter()
// {
//     // cudaGraphDestory();
//     // cudaGraphExecDestory();
//     // cudaStreamDestory();
// }